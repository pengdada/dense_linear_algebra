#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <cstdlib>
#include <vector>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "sgemm_cuda_kernel.h"
#include "sgemm_cuda_kernel.cu"
#include "dev_array.h"
#include <math.h>

using namespace std;

int main(int argc, char **argv)
{
    int N;

    if (argc !=2)  {
	printf("Usage: ./1_dense_cuda <matrix_size> \n");
	return 1;
    } else {
	N = atoi(argv[1]);
    }

    // Perform matrix multiplication C = A*B
    // where A, B and C are NxN matrices
    int SIZE = N*N;

    hipEvent_t start, stop;

    // create cuda timer events
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // start the timer
    hipEventRecord(start, NULL);

    // Allocate memory on the host
    vector<float> h_A(SIZE);
    vector<float> h_B(SIZE);
    vector<float> h_C(SIZE);

    // Initialize matrices on the host
    for (int i=0; i<N; i++){
        for (int j=0; j<N; j++){
            h_A[i*N+j] = sin(i);
            h_B[i*N+j] = cos(j);
        }
    }

    // Allocate memory on the device
    dev_array<float> d_A(SIZE);
    dev_array<float> d_B(SIZE);
    dev_array<float> d_C(SIZE);

    d_A.set(&h_A[0], SIZE);
    d_B.set(&h_B[0], SIZE);

    matrixMultiplication(d_A.getData(), d_B.getData(), d_C.getData(), N);
    hipDeviceSynchronize();

    d_C.get(&h_C[0], SIZE);
    hipDeviceSynchronize();

    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);

    float msec_total = 0.0f;
    hipEventElapsedTime(&msec_total, start, stop);

    // Compute and print the performance
    float msec_per_matrix_mul = msec_total;
    double flops_per_matrix_mul = 2.0 * (double)N * (double)N * (double)N;
    double giga_flops = (flops_per_matrix_mul * 1.0e-9f) / (msec_per_matrix_mul / 1000.0f);
    printf(
        "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops\n",
        giga_flops,
        msec_per_matrix_mul,
        flops_per_matrix_mul);


    float *cpu_C;
    cpu_C=new float[SIZE];

    // Now do the matrix multiplication on the CPU
    float sum;
    for (int row=0; row<N; row++){
        for (int col=0; col<N; col++){
            sum = 0.f;
            for (int n=0; n<N; n++){
                sum += h_A[row*N+n]*h_B[n*N+col];
            }
            cpu_C[row*N+col] = sum;
        }
    }

    double err = 0;
    // Check the result and make sure it is correct
    for (int ROW=0; ROW < N; ROW++){
        for (int COL=0; COL < N; COL++){
            err += cpu_C[ROW * N + COL] - h_C[ROW * N + COL];
        }
    }

    cout << "Error: " << err << endl;

    return 0;
}